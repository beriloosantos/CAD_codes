#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define CHECK_CUDA(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error: %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

#define CHECK_CUSOLVER(call) \
    do { \
        hipsolverStatus_t err = call; \
        if (err != HIPSOLVER_STATUS_SUCCESS) { \
            fprintf(stderr, "cuSOLVER error: %d at %s:%d\n", err, __FILE__, __LINE__); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

// Normalize eigenvectors and adjust sign
void normalizeEigenvectors(double* eigenvectors, int n) {
    for (int j = 0; j < n; j++) {
        // Find the largest component (in absolute value) for the current eigenvector
        double maxVal = 0.0;
        int maxIndex = 0;
        for (int i = 0; i < n; i++) {
            if (fabs(eigenvectors[i + j * n]) > fabs(maxVal)) {
                maxVal = eigenvectors[i + j * n];
                maxIndex = i;
            }
        }

        // Flip the sign of the eigenvector if the largest component is negative
        if (maxVal < 0) {
            for (int i = 0; i < n; i++) {
                eigenvectors[i + j * n] = -eigenvectors[i + j * n];
            }
        }

        // Normalize the eigenvector to have unit length
        double norm = 0.0;
        for (int i = 0; i < n; i++) {
            norm += eigenvectors[i + j * n] * eigenvectors[i + j * n];
        }
        norm = sqrt(norm);

        for (int i = 0; i < n; i++) {
            eigenvectors[i + j * n] /= norm;
        }
    }
}

int main() {
    const int n = 3; // Size of the matrices (n x n)

    // Input matrices A and B (symmetric, column-major order)
    double A[] = {4.0, 1.0, 1.0,
                  1.0, 2.0, 0.0,
                  1.0, 0.0, 3.0};
    double B[] = {2.0, 0.0, 0.0,
                  0.0, 3.0, 0.0,
                  0.0, 0.0, 1.0};

    // Output eigenvalues and eigenvectors
    double* eigenvalues = (double*)malloc(n * sizeof(double));
    double* eigenvectors = (double*)malloc(n * n * sizeof(double));

    // Device memory pointers
    double *d_A, *d_B, *d_eigenvalues, *d_work;
    int *d_info;
    int lwork = 0;

    // Create cuSOLVER handle
    hipsolverHandle_t handle;
    CHECK_CUSOLVER(hipsolverDnCreate(&handle));

    // Allocate device memory
    CHECK_CUDA(hipMalloc((void**)&d_A, n * n * sizeof(double)));
    CHECK_CUDA(hipMalloc((void**)&d_B, n * n * sizeof(double)));
    CHECK_CUDA(hipMalloc((void**)&d_eigenvalues, n * sizeof(double)));
    CHECK_CUDA(hipMalloc((void**)&d_info, sizeof(int)));

    // Copy matrices A and B to device
    CHECK_CUDA(hipMemcpy(d_A, A, n * n * sizeof(double), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, B, n * n * sizeof(double), hipMemcpyHostToDevice));

    // Query workspace size
    CHECK_CUSOLVER(hipsolverDnDsygvd_bufferSize(
        handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_LOWER,
        n, d_A, n, d_B, n, d_eigenvalues, &lwork));

    // Allocate workspace
    CHECK_CUDA(hipMalloc((void**)&d_work, lwork * sizeof(double)));

    // Solve generalized eigenvalue problem
    CHECK_CUSOLVER(hipsolverDnDsygvd(
        handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_LOWER,
        n, d_A, n, d_B, n, d_eigenvalues, d_work, lwork, d_info));

    // Copy results back to host
    CHECK_CUDA(hipMemcpy(eigenvalues, d_eigenvalues, n * sizeof(double), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(eigenvectors, d_A, n * n * sizeof(double), hipMemcpyDeviceToHost));

    // Normalize and adjust sign of eigenvectors
    normalizeEigenvectors(eigenvectors, n);

    // Print eigenvalues
    printf("Eigenvalues: ");
    for (int i = 0; i < n; i++) {
        printf("%f ", eigenvalues[i]);
    }
    printf("\n");

    // Print eigenvectors (column-major)
    printf("Eigenvectors (column-major):\n");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("%f ", eigenvectors[i + j * n]);
        }
        printf("\n");
    }

    // Cleanup
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_eigenvalues));
    CHECK_CUDA(hipFree(d_work));
    CHECK_CUDA(hipFree(d_info));
    CHECK_CUSOLVER(hipsolverDnDestroy(handle));

    free(eigenvalues);
    free(eigenvectors);

    return 0;
}